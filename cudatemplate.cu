
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

using namespace std;

#define MIN(a,b) ({ \
    __typeof__ (a) _a = (a); \
    __typeof__ (b) _b = (b); \
    _a < _b ? _a : _b; })

#define MAX(a,b) ({ \
    __typeof__ (a) _a = (a); \
    __typeof__ (b) _b = (b); \
    _a > _b ? _a : _b; })

#define DIV(a,b) \
    ({((a) % (b) == 0) ? ((a) / (b)) : ((a) / (b) + 1); })

#define CUDA_SAFE_CALL_NO_SYNC(call) do {                                \
    hipError_t err = call;                                                \
    if (hipSuccess != err) {                                            \
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",    \
                __FILE__, __LINE__, hipGetErrorString(err));            \
        exit(EXIT_FAILURE);                                              \
    } } while(0)

#define HOST_CHECK_POINTER(p) ({                                         \
    __typeof__ (p) __HOST_TEMP_POINTER = (p);                            \
    (__HOST_TEMP_POINTER == NULL) ? ({                                   \
        fprintf(stderr, "malloc error in file '%s' in line %i.\n",       \
                __FILE__, __LINE__);                                     \
        exit(EXIT_FAILURE);                                              \
        __HOST_TEMP_POINTER;                                             \
    }) :                                                                 \
    __HOST_TEMP_POINTER; })


__global__ void kernelTemplate(int *array, int arrayLength) {
    __shared__ int cache[1024];

    unsigned int offset = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (offset < arrayLength) {
        cache[threadIdx.x] = array[offset];
    }
    __syncthreads();

    cache[threadIdx.x] ++;

    if (offset < arrayLength) {
        array[offset] = cache[threadIdx.x];
    }
}

int main(int argc, char **argv) {
    int array_length = 10000;
    
    int *array = HOST_CHECK_POINTER((int *)malloc(array_length * sizeof(int)));
    memset(array, 0, sizeof(array));

    int *dev_array;
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&dev_array, array_length * sizeof(int)));
    
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(dev_array, array, array_length * sizeof(int), hipMemcpyHostToDevice));

    dim3 blocksPerGrid(DIV(array_length, 1024));
    dim3 threadsPerBlock(1024);

    hipEvent_t timer_start, timer_end;
    CUDA_SAFE_CALL_NO_SYNC(hipEventCreate(&timer_start));
    CUDA_SAFE_CALL_NO_SYNC(hipEventCreate(&timer_end));
    CUDA_SAFE_CALL_NO_SYNC(hipEventRecord(timer_start, 0));
    
    kernelTemplate<<<blocksPerGrid, threadsPerBlock>>>(dev_array, array_length);
    CUDA_SAFE_CALL_NO_SYNC(hipPeekAtLastError());  
    CUDA_SAFE_CALL_NO_SYNC(hipDeviceSynchronize());

    CUDA_SAFE_CALL_NO_SYNC(hipEventRecord(timer_end, 0));
    CUDA_SAFE_CALL_NO_SYNC(hipEventSynchronize(timer_end));

    float timer_elapsed;
    CUDA_SAFE_CALL_NO_SYNC(hipEventElapsedTime(&timer_elapsed, timer_start, timer_end));
    printf ("Time: %3.1f ms\n", timer_elapsed);
    
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(array, dev_array, array_length * sizeof(int), hipMemcpyDeviceToHost));
    
    int flag = 0;
    for (int i = 0; i < array_length; i++) {
        if (array[i] != 1) {
            printf ("kernel failed\n");
            flag ++;
            break;
        }
    }

    if (flag == 0)
        printf ("kernel sucessed\n");
    
    free(array);
    CUDA_SAFE_CALL_NO_SYNC(hipFree(dev_array));
    
    return 0;
}
